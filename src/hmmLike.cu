#include "hip/hip_runtime.h"
#include "hmmLike.hcu"

using namespace std;

namespace HMMLikeCUDA {

// basically avoid singularity or floating point error
__constant__ float norm;

__device__ void UnpackGLs(char GLset, float (&GLs)[3]) {

  GLs[0] = (((GLset >> 4) & 15) + 0.5f) / 16;
  GLs[1] = ((GLset & 15) + 0.5f) / 16;
  GLs[2] = max(1 - GLs[0] - GLs[1], 0.0f);
}

__device__ void FillEmit(const float (&GLs)[3], float (&emit)[4]) {

  for (unsigned i = 0; i < 4; ++i)
    emit[i] = mutationMat[i] * GLs[0] + mutationMat[i + 4 * 1] * GLs[1] +
              mutationMat[i + 4 * 2] * GLs[1] + mutationMat[i + 4 * 3] * GLs[2];
}

// test if bit I is 1
__device__ uint64_t test(const uint64_t *P, unsigned I) {
  return (P[I >> WordShift] >> (I & WordMod)) & static_cast<uint64_t>(1);
}

__device__ float hmmLike(unsigned idx, const unsigned (&hapIdxs)[4],
                         const char *__restrict__ d_packedGLs,
                         unsigned packedGLStride,
                         const uint64_t *__restrict__ d_hapPanel) {

  // pull the four haplotypes into f0, f1, m0 and m1
  const uint64_t *f0 = &d_hapPanel[hapIdxs[0] * WN],
                 *f1 = &d_hapPanel[hapIdxs[1] * WN],
                 *m0 = &d_hapPanel[hapIdxs[2] * WN],
                 *m1 = &d_hapPanel[hapIdxs[3] * WN];

  // ##########
  // Convert packed GLs back to floats

  float GLs[3];
  UnpackGLs(d_packedGLs[idx], GLs);

  // pull out phase emission and transition probabilities
  float emit[4];
  FillEmit(GLs, emit);

  float sum, score = 0;

  // l00 = prob of 0|0 phase, etc.
  // all set to 1/4 * emission probability
  float l00 = 0.25f * emit[(test(f0, 0) << 1) | test(m0, 0)],
        l01 = 0.25f * emit[(test(f0, 0) << 1) | test(m1, 0)];
  float l10 = 0.25f * emit[(test(f1, 0) << 1) | test(m0, 0)],
        l11 = 0.25f * emit[(test(f1, 0) << 1) | test(m1, 0)];

  for (int site = 1; site < NUMSITES; ++site) {
    // move to next site for e and t

    // #########
    // Convert packed GLs back to floats

    UnpackGLs(d_packedGLs[idx + site * packedGLStride], GLs);

    // fill emit with next site's emission matrix
    FillEmit(GLs, emit);

    // bxx = backward probabilities of being in phase xx
    const float b00 = l00 * transitionMat[site * 3] +
                      (l01 + l10) * transitionMat[site * 3 + 1] +
                      l11 * transitionMat[site * 3 + 2];
    const float b01 = l01 * transitionMat[site * 3] +
                      (l00 + l11) * transitionMat[site * 3 + 1] +
                      l10 * transitionMat[site * 3 + 2];
    const float b10 = l10 * transitionMat[site * 3] +
                      (l00 + l11) * transitionMat[site * 3 + 1] +
                      l01 * transitionMat[site * 3 + 2];
    const float b11 = l11 * transitionMat[site * 3] +
                      (l01 + l10) * transitionMat[site * 3 + 1] +
                      l00 * transitionMat[site * 3 + 2];

    l00 = b00 * emit[(test(f0, site) << 1) | test(m0, site)];
    l01 = b01 * emit[(test(f0, site) << 1) | test(m1, site)];
    l10 = b10 * emit[(test(f1, site) << 1) | test(m0, site)];
    l11 = b11 * emit[(test(f1, site) << 1) | test(m1, site)];

    // rescale probabilities if they become too small
    // hopefully this does not happen too often...
    if ((sum = l00 + l01 + l10 + l11) < norm) {
      sum = 1.0f / sum;
      score -= logf(sum); // add sum to score
      l00 *= sum;
      l01 *= sum;
      l10 *= sum;
      l11 *= sum;
    }
  }

  return score + logf(l00 + l01 + l10 + l11);
};

// definition of HMM kernel
__global__ void findHapSet(const char *__restrict__ d_packedGLs,
                           const uint64_t *__restrict__ d_hapPanel,
                           const unsigned *__restrict__ d_hapIdxs,
                           const unsigned *__restrict__ d_extraPropHaps,
                           unsigned *d_chosenHapIdxs, unsigned numSamples,
                           unsigned numCycles, hiprandStateXORWOW_t *globalState
#ifdef DEBUG
                           ,
                           float *d_likes
#endif
                           ) {

  const float S = 1;
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  if (idx < numSamples) {

    hiprandStateXORWOW_t localState = globalState[idx];

    unsigned hapIdxs[4];
    for (int i = 0; i < 4; ++i)
      hapIdxs[i] = d_hapIdxs[idx + numSamples * i];

    // define emission matrix
    float curr = hmmLike(idx, hapIdxs, d_packedGLs, numSamples, d_hapPanel);

#ifdef DEBUG
    // debugging ...
    if (idx == 0)
      d_likes[0] = curr;
#endif

    // pick a random haplotype to replace with another one from all
    // haplotypes.  calculate the new probability of the model given
    // those haplotypes.
    // accept new set if probability has increased.
    // otherwise, accept with penalized probability

    for (int cycle = 0; cycle < numCycles; ++cycle) {

      // replace a sample
      unsigned replaceHapNum = hiprand(&localState) & 3;
      unsigned origHap = hapIdxs[replaceHapNum];

      hapIdxs[replaceHapNum] = d_extraPropHaps[idx + cycle * numSamples];
      float prop = hmmLike(idx, hapIdxs, d_packedGLs, numSamples, d_hapPanel);

      // accept new set
      if (hiprand_uniform(&localState) < expf((prop - curr) * S))
        curr = prop;
      // reject new set
      else
        hapIdxs[replaceHapNum] = origHap;

#ifdef DEBUG
      if (idx == 0)
        d_likes[cycle + 1] = curr;
#endif

    } // last of numCycles
    for (int i = 0; i < 4; ++i)
      d_chosenHapIdxs[idx + numSamples * i] = hapIdxs[i];

    // update global state
    globalState[idx] = localState;

    // return nothing.  d_chosenHapIdxs is the return data
    return;
  }
}

// initializes random number generator states
__global__ void setup_generators(hiprandStateXORWOW_t *state, size_t stateSize,
                                 unsigned long seed) {

  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  if (idx < stateSize)
    hiprand_init(seed, idx, 0, &state[idx]);
}

void CheckDevice() {

  hipError_t err;
  err = hipGetLastError();
  if (err != hipSuccess) {
    cerr << "Error before starting: " << hipGetErrorString(err) << "\n";

    exit(EXIT_FAILURE);
  }

  int deviceCount = 0;

  // note your project will need to link with cuda.lib files on windows
  cout << "Querrying CUDA Device(s)...\n\n";

  err = hipGetDeviceCount(&deviceCount);

  if (err != hipSuccess) {
    cout << "hipGetDeviceCount returned " << err << "\n";
    cout << "Result = FAIL\n";
    exit(EXIT_FAILURE); //
  }

  cout << "Found " << deviceCount << " device(s)" << endl;

  for (int i = 0; i < deviceCount; ++i) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    cout << "Device number: " << i << endl;
    cout << "Name: " << prop.name << endl;
    cout << "Compute capability: " << prop.major << '.' << prop.minor << endl;

    if (prop.major < 3 || prop.major == 3 && prop.minor < 5) {
      cerr << "Compute capability >= 3.5 required" << endl;
      exit(EXIT_FAILURE);
    }

    // set device to use more L1 cache than shared mem
    cout << "\nSetting device to prefer L1 cache over shared mem\n";
    if (hipDeviceSetCacheConfig(hipFuncCachePreferL1) != hipSuccess) {
      cerr << "Could not set device caching preference to L1\n";
    }

    hipFuncCache_t pCacheConfig;
    if (hipDeviceGetCacheConfig(&pCacheConfig) != hipSuccess) {
      cerr << "Could not get device caching preference\n";
    }
    if (pCacheConfig == hipFuncCachePreferL1)
      cout << "Device caching preference is set to prefer L1" << endl;
    else
      cout << "Device caching preference is not set to prefer L1" << endl;
  }

  // also, this looks like as good of a place as any to define some constants
  float localNorm = powf(FLT_MIN, 2.0f / 3.0f);
  err = hipMemcpyToSymbol(HIP_SYMBOL(norm), &localNorm, sizeof(float), 0,
                           hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    cerr << "Error copying value to symbol: " << hipGetErrorString(err)
         << "\n";

    exit(EXIT_FAILURE);
  }

  return;
}

hipError_t CopyTranToDevice(const vector<float> &tran) {

  assert(tran.size() == NUMSITES * 3);
  // first three values of tran are never used
  for (unsigned i = 3; i < tran.size(); i += 3)
    assert(abs(tran[i] + 2 * tran[i + 1] + tran[i + 2] - 1) < 0.1);
  return hipMemcpyToSymbol(HIP_SYMBOL(transitionMat), tran.data(),
                            sizeof(float) * NUMSITES * 3, 0,
                            hipMemcpyHostToDevice);
}

hipError_t CopyMutationMatToDevice(const float (*mutMat)[4][4]) {

  vector<float> h_mutMat(4 * 4);
  for (int i = 0; i < 4; ++i)
    for (int j = 0; j < 4; ++j)
      h_mutMat[i + 4 * j] = ((*mutMat))[i][j];

  return hipMemcpyToSymbol(HIP_SYMBOL(mutationMat), h_mutMat.data(), sizeof(float) * 4 * 4,
                            0, hipMemcpyHostToDevice);
}

void Cleanup() { assert(hipDeviceReset() == hipSuccess); }

void RunHMMOnDevice(const thrust::device_vector<char> &d_packedGLs,
                    const vector<uint64_t> &hapPanel,
                    const vector<unsigned> &extraPropHaps, unsigned numSites,
                    unsigned numSamples, unsigned numCycles,
                    vector<unsigned> &hapIdxs, unsigned long seed) {
  assert(numSites == NUMSITES);
  assert(d_packedGLs.size() == numSites * numSamples);
  assert(hapPanel.size() >=
         WN * (*max_element(extraPropHaps.begin(), extraPropHaps.end()) + 1));
  assert(hapIdxs.size() == numSamples * 4);
  assert(extraPropHaps.size() == numSamples * numCycles);

/*
  copy packedGLs to device memory
*/
/* hopefully we don't need this code anymore...
#ifdef DEBUG
cout << "[HMMLikeCUDA] Copying packed GLs to device\n";
#endif

size_t glSize = packedGLs.size() * sizeof(char);

// allocate memory on device
char *d_packedGLs;
err = hipMalloc(&d_packedGLs, glSize);
if (err != hipSuccess) {
  cerr << "Failed to allocate packed GLs on device\n";
  exit(EXIT_FAILURE);
}

// copy data across
err =
    hipMemcpy(d_packedGLs, packedGLs.data(), glSize, hipMemcpyHostToDevice);
if (err != hipSuccess) {
  cerr << "Failed to copy packed GLs to device\n";
  exit(EXIT_FAILURE);
}
*/

#ifdef DEBUG
  // debug info
  cout << "Hap Idxs before sampling: ";
  for (int i = 0; i < hapIdxs.size(); ++i)
    cout << hapIdxs[i] << " ";
  cout << endl;
#endif

  hipError_t err = hipSuccess;
  /*
    Set up numSamp random generator states
   */
  hiprandStateXORWOW_t *devStates;
  hipMalloc(&devStates, numSamples * sizeof(hiprandStateXORWOW_t));

  // set up generators
  unsigned threadsPerBlock = 32;
  unsigned blocksPerRun = (numSamples + threadsPerBlock - 1) / threadsPerBlock;

  setup_generators << <blocksPerRun, threadsPerBlock>>>
      (devStates, numSamples, seed);
  err = hipGetLastError();
  if (err != hipSuccess) {
    cerr << "Failed to set up random states kernel: " << hipGetErrorString(err)
         << "\n";
    exit(EXIT_FAILURE);
  }

/*
  copy haplotypes to device memory
*/
#ifdef DEBUG
  cout << "[HMMLikeCUDA] Copying HapPanel to device\n";
#endif
  size_t hapPanelSize = hapPanel.size() * sizeof(uint64_t);

  // allocate memory on device
  uint64_t *d_hapPanel;
  err = hipMalloc(&d_hapPanel, hapPanelSize);
  if (err != hipSuccess) {
    cerr << "Failed to allocate hapPanel on device\n";
    exit(EXIT_FAILURE);
  }

  // copy data across
  err = hipMemcpy(d_hapPanel, hapPanel.data(), hapPanelSize,
                   hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    cerr << "Failed to copy hapPanel to device\n";
    exit(EXIT_FAILURE);
  }

/*
  copy initial hap indices to device memory
*/
#ifdef DEBUG
  cout << "[HMMLikeCUDA] Copying hap indices to device\n";
#endif
  size_t hapIdxsSize = hapIdxs.size() * sizeof(unsigned);

  // allocate memory on device
  unsigned *d_hapIdxs;
  err = hipMalloc(&d_hapIdxs, hapIdxsSize);
  if (err != hipSuccess) {
    cerr << "Failed to allocate hap indices on device\n";
    exit(EXIT_FAILURE);
  }

  // copy data across
  err = hipMemcpy(d_hapIdxs, hapIdxs.data(), hapIdxsSize,
                   hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    cerr << "Failed to copy hap indices to device\n";
    exit(EXIT_FAILURE);
  }

/*
  copy extra proposal haps to device memory
*/
#ifdef DEBUG
  cout << "[HMMLikeCUDA] Copying extra proposal haps to device\n";
#endif
  size_t extraPropHapsSize = extraPropHaps.size() * sizeof(unsigned);

  // allocate memory on device
  unsigned *d_extraPropHaps;
  err = hipMalloc(&d_extraPropHaps, extraPropHapsSize);
  if (err != hipSuccess) {
    cerr << "Failed to allocate extra prop haps on device\n";
    exit(EXIT_FAILURE);
  }

  // copy data across
  err = hipMemcpy(d_extraPropHaps, extraPropHaps.data(), extraPropHapsSize,
                   hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    cerr << "Failed to copy extra prop haps to device\n";
    exit(EXIT_FAILURE);
  }

  /*
    allocate device memory for results
  */
  unsigned *d_chosenHapIdxs;
  err = hipMalloc(&d_chosenHapIdxs, hapIdxsSize);
  if (err != hipSuccess) {
    cerr << "Failed to allocate memory for result hap idxs on device\n";
    exit(EXIT_FAILURE);
  }

#ifdef DEBUG
  /*
    allocate device memory for debugging floats
  */
  thrust::device_vector<float> d_likes(numCycles + 1);
  float *d_likePtr = thrust::raw_pointer_cast(d_likes.data());
#endif

  // determine thread and block size
  threadsPerBlock = 32;
  blocksPerRun = (numSamples + threadsPerBlock - 1) / threadsPerBlock;
#ifdef DEBUG
  cout << "[HMMLikeCUDA] Running with " << threadsPerBlock
       << " threads per block in " << blocksPerRun << " thread blocks\n";
#endif

  /*
    convert gd_packedGLs to raw ptr
  */
  const char *d_packedGLPtr =  thrust::raw_pointer_cast(d_packedGLs.data());

  /*
    run kernel
  */
  findHapSet << <blocksPerRun, threadsPerBlock>>>
      (d_packedGLPtr, d_hapPanel, d_hapIdxs, d_extraPropHaps, d_chosenHapIdxs,
       numSamples, numCycles, devStates
#ifdef DEBUG
       ,
       d_likePtr
#endif
       );
  hipDeviceSynchronize();
  err = hipGetLastError();
  if (err != hipSuccess) {
    cerr << "Failed to run HMM kernel: " << hipGetErrorString(err) << "\n";
    exit(EXIT_FAILURE);
  }

  /*
    copy result hap indices back to host into hapIdxs
  */
  err = hipMemcpy(hapIdxs.data(), d_chosenHapIdxs, hapIdxsSize,
                   hipMemcpyDeviceToHost);
  if (err != hipSuccess) {
    cerr << "Failed to copy chosen indices to host: " << hipGetErrorString(err)
         << "\nCode: " << err << "\n";
    exit(EXIT_FAILURE);
  }

  assert(hipFree(devStates) == hipSuccess);
  assert(hipFree(d_chosenHapIdxs) == hipSuccess);
  assert(hipFree(d_extraPropHaps) == hipSuccess);
  assert(hipFree(d_hapIdxs) == hipSuccess);
  assert(hipFree(d_hapPanel) == hipSuccess);

#ifdef DEBUG
  thrust::host_vector<float> h_likes(numCycles + 1);
  h_likes = d_likes;

  // debug info
  cout << "Hap Idxs after sampling: ";
  for (int i = 0; i < hapIdxs.size(); ++i)
    cout << hapIdxs[i] << " ";
  cout << endl;

  cout << "cycle likelihoods: ";
  for (int i = 0; i < numCycles + 1; ++i)
    cout << h_likes[i] << " ";
  cout << endl;
#endif

  // return nothing as the return data is stored in hapIdxs
  return;
}
}
