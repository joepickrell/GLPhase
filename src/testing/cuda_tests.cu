#include "hip/hip_runtime.h"

// cuda does not seem to like any of these in header files...

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include "hmmLike.hcu"

using namespace std;

namespace HMMLikeCUDATest {

__global__ void GlobalFillEmit(const float *GLs, float *emit) {
  float nEmit[4];
  float nGLs[3];
  for (int i = 0; i < 3; ++i)
    nGLs[i] = GLs[i];
  HMMLikeCUDA::FillEmit(nGLs, nEmit);
  for (int i = 0; i < 4; ++i)
    emit[i] = nEmit[i];
}

void FillEmit(const vector<float> &GLs, vector<float> &emit) {

  assert(GLs.size() == 3);
  assert(emit.size() == 4);
  hipError_t err = hipSuccess;
  float *d_GLs;
  err = hipMalloc(&d_GLs, 3 * sizeof(float));
  assert(err == hipSuccess);

  err =
      hipMemcpy(d_GLs, GLs.data(), 3 * sizeof(float), hipMemcpyHostToDevice);
  assert(err == hipSuccess);

  float *d_emit;
  err = hipMalloc(&d_emit, 4 * sizeof(float));
  assert(err == hipSuccess);

  GlobalFillEmit << <1, 1>>> (d_GLs, d_emit);

  err = hipMemcpy(emit.data(), d_emit, 4 * sizeof(float),
                   hipMemcpyDeviceToHost);
  assert(err == hipSuccess);
};

__global__ void GlobalHmmLike(unsigned idx, const unsigned (*hapIdxs)[4],
                              const uint32_t *__restrict__ d_packedGLs,
                              unsigned packedGLStride,
                              const uint64_t *__restrict__ d_hapPanel,
                              const float *__restrict__ d_codeBook,
                              float *retLike) {
  *retLike = HMMLikeCUDA::hmmLike(idx, *hapIdxs, d_packedGLs, packedGLStride,
                                  d_hapPanel, d_codeBook);

  return;
}

float CallHMMLike(unsigned idx, const unsigned (*hapIdxs)[4],
                  unsigned packedGLStride, const vector<uint64_t> &h_hapPanel) {

  hipError_t err = hipSuccess;

  /*
  copy initial hap indices to device memory
*/
  cout << "Copying hap indices to device\n";
  // allocate memory on device
  unsigned(*d_hapIdxs)[4];
  err = hipMalloc(&d_hapIdxs, 4 * sizeof(unsigned));
  if (err != hipSuccess) {
    cerr << "Failed to allocate\n";
    exit(EXIT_FAILURE);
  }

  // copy data across
  err = hipMemcpy(d_hapIdxs, hapIdxs, 4 * sizeof(unsigned),
                   hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    cerr << "Failed to copy\n";
    exit(EXIT_FAILURE);
  }

  /*
  convert gd_packedGLs to raw ptr
*/
  assert(HMMLikeCUDA::gd_packedGLs);
  const uint32_t *d_packedGLPtr =
      thrust::raw_pointer_cast(HMMLikeCUDA::gd_packedGLs->data());

  assert(HMMLikeCUDA::gd_codeBook);
  const float *d_codeBookPtr =
      thrust::raw_pointer_cast(HMMLikeCUDA::gd_codeBook->data());

  /*
    copy haplotypes to device memory
  */
  cout << "Copying HapPanel to device\n";
  size_t hapPanelSize = h_hapPanel.size() * sizeof(uint64_t);

  // allocate memory on device
  uint64_t *d_hapPanel;
  err = hipMalloc(&d_hapPanel, hapPanelSize);
  if (err != hipSuccess) {
    cerr << "Failed to allocate hapPanel on device\n";
    exit(EXIT_FAILURE);
  }

  // copy data across
  err = hipMemcpy(d_hapPanel, h_hapPanel.data(), hapPanelSize,
                   hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    cerr << "Failed to copy hapPanel to device\n";
    exit(EXIT_FAILURE);
  }

  // create result on device
  thrust::device_vector<float> d_like(1, 1);
  float *d_likePtr = thrust::raw_pointer_cast(&d_like[0]);
  /*
    run kernel
  */
  GlobalHmmLike << <1, 1>>> (idx, d_hapIdxs, d_packedGLPtr, packedGLStride,
                             d_hapPanel, d_codeBookPtr, d_likePtr);
  hipDeviceSynchronize();
  err = hipGetLastError();
  if (err != hipSuccess) {
    cerr << "Failed to run HMMLike kernel: " << hipGetErrorString(err) << "\n";
    exit(EXIT_FAILURE);
  }

  thrust::host_vector<float> h_like = d_like;
  return d_like[0];
};

__global__ void GlobalUnpackGLsWithCodeBook(uint32_t GLcodes, float *GLs,
                                            const float *__restrict__ codeBook,
                                            unsigned char glIdx) {
  float nGLs[3];
  HMMLikeCUDA::UnpackGLsWithCodeBook(GLcodes, nGLs, codeBook, glIdx);
  for (int i = 0; i < 3; ++i)
    GLs[i] = nGLs[i];
}

void UnpackGLsWithCodeBook(uint32_t GLcodes, vector<float> &GLs,
                           unsigned char glIdx) {
  assert(GLs.size() == 3);

  thrust::device_vector<float> d_GLs(3, 0);
  float *d_GLPtr = thrust::raw_pointer_cast(d_GLs.data());
  const float *d_codeBook =
      thrust::raw_pointer_cast(HMMLikeCUDA::gd_codeBook->data());

  GlobalUnpackGLsWithCodeBook << <1, 1>>> (GLcodes, d_GLPtr, d_codeBook, glIdx);

  thrust::host_vector<float> h_GLs;
  h_GLs = d_GLs;
  for (int i = 0; i < 3; ++i)
    GLs[i] = h_GLs[i];
}

hipError_t CopyTranToHost(vector<float> &tran) {

  assert(tran.size() == NUMSITES * 3);
  return hipMemcpyFromSymbol(tran.data(), HMMLikeCUDA::transitionMat,
                              sizeof(float) * NUMSITES * 3, 0,
                              hipMemcpyDeviceToHost);
}

hipError_t CopyMutMatToHost(vector<float> &mutMat) {

  assert(mutMat.size() == 4 * 4);
  return hipMemcpyFromSymbol(mutMat.data(), HMMLikeCUDA::mutationMat,
                              sizeof(float) * 4 * 4, 0, hipMemcpyDeviceToHost);
}

__global__ void GetRNs(unsigned *d_fillRNs, hiprandStateMtgp32 *globalState,
                       size_t numRNs) {

    int idx = blockDim.x * blockIdx.x + threadIdx.x;
  for (size_t i = 0; i != numRNs; ++i)
    d_fillRNs[i+idx * numRNs] = hiprand(&globalState[blockIdx.x]);
  return;
}

void FillRNs(thrust::host_vector<unsigned> &h_rns, size_t numRNs) {

  thrust::device_vector<unsigned> d_rns;
  d_rns.resize(numRNs, 0);
  unsigned *d_rnsPtr = thrust::raw_pointer_cast(d_rns.data());
  assert(HMMLikeCUDA::gd_devMTGPStates);
  GetRNs << <1, 2>>> (d_rnsPtr, HMMLikeCUDA::gd_devMTGPStates, numRNs/2);

  h_rns = d_rns;
  return;
}
}
