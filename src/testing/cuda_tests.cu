#include "hip/hip_runtime.h"
//#include "hip/hip_runtime.h"
//#include "hip/hip_runtime.h"
#include "hmmLike.hcu"
#include <iostream>

using namespace std;
using namespace HMMLikeCUDA;

namespace HMMLikeCUDATest {
__global__ void HostUnpackGLs(char GLset, float *GLs) { UnpackGLs(GLset, GLs); }

bool UnpackGLs(char GLset, float *GLs) {

  hipError_t err = hipSuccess;

  // figure out how big output will be
  size_t size = 3 * sizeof(float);

  // Allocate the device input GLset
  float *d_GLs = NULL;
  err = hipMalloc((void **)&d_GLs, size);

  if (err != hipSuccess) {
    cerr << "Failed to allocate device GLset vector (error code "
         << hipGetErrorString(err) << ")!\n";

    exit(EXIT_FAILURE);
  }

  HostUnpackGLs << <1, 1>>> (GLset, d_GLs);

  const hipError_t retErr = hipGetLastError();

  err = hipMemcpy(GLs, d_GLs, size, hipMemcpyDeviceToHost);

  if (err != hipSuccess) {
    cerr << "Failed to copy vector GLs from device to host (error code "
         << hipGetErrorString(err) << ")!\n";

    exit(EXIT_FAILURE);
  }

  // deallocate memory
  hipFree(d_GLs);

  if (retErr == hipSuccess)
    return true;
  else
    return false;
}

hipError_t CopyTranToHost(vector<float> &tran) {

  assert(tran.size() == NUMSITES * 3);
  return hipMemcpyFromSymbol(tran.data(), transitionMat,
                              sizeof(float) * NUMSITES * 3, 0,
                              hipMemcpyDeviceToHost);
}
hipError_t CopyMutMatToHost(vector<float> &mutMat) {

  assert(mutMat.size() == 4 * 4);
  return hipMemcpyFromSymbol(mutMat.data(), mutationMat, sizeof(float) * 4 * 4,
                              0, hipMemcpyDeviceToHost);
}
}
