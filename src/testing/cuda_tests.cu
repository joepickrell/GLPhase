#include "hip/hip_runtime.h"
//#include "hip/hip_runtime.h"
//#include "hip/hip_runtime.h"
#include "hmmLike.hcu"

using namespace std;
using namespace HMMLikeCUDA;

namespace HMMLikeCUDATest {

__global__ void GlobalFillEmit(const float *GLs, float *emit) {
  float nEmit[4];
  float nGLs[3];
  for (int i = 0; i < 3; ++i)
    nGLs[i] = GLs[i];
  FillEmit(nGLs, nEmit);
  for (int i = 0; i < 4; ++i)
    emit[i] = nEmit[i];
}

void FillEmit(const vector<float> &GLs, vector<float> &emit) {

  assert(GLs.size() == 3);
  assert(emit.size() == 4);
  hipError_t err = hipSuccess;
  float *d_GLs;
  err = hipMalloc(&d_GLs, 3 * sizeof(float));
  assert(err == hipSuccess);

  err =
      hipMemcpy(d_GLs, GLs.data(), 3 * sizeof(float), hipMemcpyHostToDevice);
  assert(err == hipSuccess);

  float *d_emit;
  err = hipMalloc(&d_emit, 4 * sizeof(float));
  assert(err == hipSuccess);

  GlobalFillEmit << <1, 1>>> (d_GLs, d_emit);

  err = hipMemcpy(emit.data(), d_emit, 4 * sizeof(float),
                   hipMemcpyDeviceToHost);
  assert(err == hipSuccess);
};

__global__ void GlobalHmmLike(unsigned idx, const unsigned (*hapIdxs)[4],
                              const char *__restrict__ d_packedGLs,
                              unsigned packedGLStride,
                              const uint64_t *__restrict__ d_hapPanel,
                              float *retLike) {
  *retLike = hmmLike(idx, *hapIdxs, d_packedGLs, packedGLStride, d_hapPanel);

  return;
}

float CallHMMLike(unsigned idx, const unsigned (*hapIdxs)[4],
                  const vector<char> &packedGLs, unsigned packedGLStride,
                  const vector<uint64_t> &h_hapPanel) {

  hipError_t err = hipSuccess;

  /*
  copy initial hap indices to device memory
*/
  cout << "Copying hap indices to device\n";
  // allocate memory on device
  unsigned(*d_hapIdxs)[4];
  err = hipMalloc(&d_hapIdxs, 4 * sizeof(unsigned));
  if (err != hipSuccess) {
    cerr << "Failed to allocate\n";
    exit(EXIT_FAILURE);
  }

  // copy data across
  err = hipMemcpy(d_hapIdxs, hapIdxs, 4 * sizeof(unsigned),
                   hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    cerr << "Failed to copy\n";
    exit(EXIT_FAILURE);
  }

  /*
    copy packedGLs to device memory
  */
  cout << "Copying packed GLs to device\n";
  size_t glSize = packedGLs.size() * sizeof(char);

  // allocate memory on device
  char *d_packedGLs;
  err = hipMalloc(&d_packedGLs, glSize);
  if (err != hipSuccess) {
    cerr << "Failed to allocate packed GLs on device\n";
    exit(EXIT_FAILURE);
  }

  // copy data across
  err =
      hipMemcpy(d_packedGLs, packedGLs.data(), glSize, hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    cerr << "Failed to copy packed GLs to device\n";
    exit(EXIT_FAILURE);
  }

  /*
    copy haplotypes to device memory
  */
  cout << "Copying HapPanel to device\n";
  size_t hapPanelSize = h_hapPanel.size() * sizeof(uint64_t);

  // allocate memory on device
  uint64_t *d_hapPanel;
  err = hipMalloc(&d_hapPanel, hapPanelSize);
  if (err != hipSuccess) {
    cerr << "Failed to allocate hapPanel on device\n";
    exit(EXIT_FAILURE);
  }

  // copy data across
  err = hipMemcpy(d_hapPanel, h_hapPanel.data(), hapPanelSize,
                   hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    cerr << "Failed to copy hapPanel to device\n";
    exit(EXIT_FAILURE);
  }

  // create result on device
  thrust::device_vector<float> d_like(1, 1);
  float *d_likePtr = thrust::raw_pointer_cast(&d_like[0]);
  /*
    run kernel
  */
  GlobalHmmLike << <1, 1>>>
      (idx, d_hapIdxs, d_packedGLs, packedGLStride, d_hapPanel, d_likePtr);
  hipDeviceSynchronize();
  err = hipGetLastError();
  if (err != hipSuccess) {
    cerr << "Failed to run HMMLike kernel: " << hipGetErrorString(err) << "\n";
    exit(EXIT_FAILURE);
  }

  thrust::host_vector<float> h_like = d_like;
  return d_like[0];
};

__global__ void GlobalUnpackGLs(char GLset, float *GLs) {
  float nGLs[3];
  UnpackGLs(GLset, nGLs);
  for (int i = 0; i < 3; ++i)
    GLs[i] = nGLs[i];
}

bool UnpackGLs(char GLset, float *GLs) {

  hipError_t err = hipSuccess;

  // figure out how big output will be
  size_t size = 3 * sizeof(float);

  // Allocate the device input GLset
  float *d_GLs;
  err = hipMalloc(&d_GLs, size);

  if (err != hipSuccess) {
    cerr << "Failed to allocate device GLset vector (error code "
         << hipGetErrorString(err) << ")!\n";

    exit(EXIT_FAILURE);
  }

  GlobalUnpackGLs << <1, 1>>> (GLset, d_GLs);

  const hipError_t retErr = hipGetLastError();

  err = hipMemcpy(GLs, d_GLs, size, hipMemcpyDeviceToHost);

  if (err != hipSuccess) {
    cerr << "Failed to copy vector GLs from device to host (error code "
         << hipGetErrorString(err) << ")!\n";

    exit(EXIT_FAILURE);
  }

  // deallocate memory
  hipFree(d_GLs);

  if (retErr == hipSuccess)
    return true;
  else {
    cerr << "Failed to unpack gls. Error: " << hipGetErrorString(retErr)
         << "\nError code: " << retErr << "\n";
    exit(EXIT_FAILURE);
  }
}

hipError_t CopyTranToHost(vector<float> &tran) {

  assert(tran.size() == NUMSITES * 3);
  return hipMemcpyFromSymbol(tran.data(), transitionMat,
                              sizeof(float) * NUMSITES * 3, 0,
                              hipMemcpyDeviceToHost);
}

hipError_t CopyMutMatToHost(vector<float> &mutMat) {

  assert(mutMat.size() == 4 * 4);
  return hipMemcpyFromSymbol(mutMat.data(), mutationMat, sizeof(float) * 4 * 4,
                              0, hipMemcpyDeviceToHost);
}
}
