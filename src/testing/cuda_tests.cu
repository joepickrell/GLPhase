#include "hip/hip_runtime.h"
//#include "hip/hip_runtime.h"
//#include "hip/hip_runtime.h"
#include "hmmLike.hcu"
#include <iostream>

using namespace std;
using namespace HMMLikeCUDA;

namespace HMMLikeCUDATest {

__global__ void GlobalFillEmit(const float *GLs, float *emit) {
  float nEmit[4];
  float nGLs[3];
  for (int i = 0; i < 3; ++i)
    nGLs[i] = GLs[i];
  FillEmit(nGLs, nEmit);
  for (int i = 0; i < 4; ++i)
    emit[i] = nEmit[i];
}

extern "C" void FillEmit(const vector<float> &GLs, vector<float> &emit) {

  assert(GLs.size() == 3);
  assert(emit.size() == 4);
  hipError_t err = hipSuccess;
  float *d_GLs;
  err = hipMalloc(&d_GLs, 3 * sizeof(float));
  assert(err == hipSuccess);

  err = hipMemcpy(d_GLs, GLs.data(), 3 * sizeof(float), hipMemcpyHostToDevice);
  assert(err == hipSuccess);

  float *d_emit;
  err = hipMalloc(&d_emit, 4 * sizeof(float));
  assert(err == hipSuccess);

  GlobalFillEmit << <1, 1>>> (d_GLs, d_emit);

  err = hipMemcpy(emit.data(), d_emit, 4 * sizeof(float), hipMemcpyDeviceToHost);
  assert(err == hipSuccess);
};

__global__ void GlobalUnpackGLs(char GLset, float *GLs) {
  float nGLs[3];
  UnpackGLs(GLset, nGLs);
  for (int i = 0; i < 3; ++i)
    GLs[i] = nGLs[i];
}

extern "C" bool UnpackGLs(char GLset, float *GLs) {

  hipError_t err = hipSuccess;

  // figure out how big output will be
  size_t size = 3 * sizeof(float);

  // Allocate the device input GLset
  float *d_GLs;
  err = hipMalloc(&d_GLs, size);

  if (err != hipSuccess) {
    cerr << "Failed to allocate device GLset vector (error code "
         << hipGetErrorString(err) << ")!\n";

    exit(EXIT_FAILURE);
  }

  GlobalUnpackGLs << <1, 1>>> (GLset, d_GLs);

  const hipError_t retErr = hipGetLastError();

  err = hipMemcpy(GLs, d_GLs, size, hipMemcpyDeviceToHost);

  if (err != hipSuccess) {
    cerr << "Failed to copy vector GLs from device to host (error code "
         << hipGetErrorString(err) << ")!\n";

    exit(EXIT_FAILURE);
  }

  // deallocate memory
  hipFree(d_GLs);

  if (retErr == hipSuccess)
    return true;
  else {
    cerr << "Failed to unpack gls. Error: " << hipGetErrorString(retErr)
         << "\nError code: " << retErr << "\n";
    exit(EXIT_FAILURE);
  }
}

extern "C" hipError_t CopyTranToHost(vector<float> &tran) {

  assert(tran.size() == NUMSITES * 3);
  return hipMemcpyFromSymbol(tran.data(), transitionMat,
                              sizeof(float) * NUMSITES * 3, 0,
                              hipMemcpyDeviceToHost);
}

extern "C" hipError_t CopyMutMatToHost(vector<float> &mutMat) {

  assert(mutMat.size() == 4 * 4);
  return hipMemcpyFromSymbol(mutMat.data(), mutationMat, sizeof(float) * 4 * 4,
                              0, hipMemcpyDeviceToHost);
}
}
